#include "hip/hip_runtime.h"
#include <thrust/version.h>
#include <thrust/device_ptr.h>
#include <thrust/sort.h>

#include <cpuid.h>
#include <wmmintrin.h>
#include <emmintrin.h>

#include <cstdint>
#include <iostream>

// Macro to catch CUDA errors in CUDA runtime calls
#define CUDA_SAFE_CALL(call) \
do { \
    hipError_t err = call; \
    if (hipSuccess != err) { \
        fprintf (stderr, "Cuda error in file '%s' in line %i : %s.", \
                 __FILE__, __LINE__, hipGetErrorString(err) ); \
        exit(EXIT_FAILURE); \
    } \
} while (0)

using aes_block = uint4;

__constant__ uint CUDA_TBOX[4][256] = {
    {0,},
    {0,},
    {0,},
    {0,},
};

__shared__ uint TBOX[4][256];
__device__ aes_block CUDA_ROUND_KEYS[2][15];
__shared__ aes_block ROUND_KEYS[2][15];

__device__ inline aes_block aes_enc(uint64_t inp, const int key) {
    aes_block state = {0,0,__brev(inp >> 32ull),__brev((uint)inp)};
    state.x ^= ROUND_KEYS[key][0].x;
    state.y ^= ROUND_KEYS[key][0].y;
    state.z ^= ROUND_KEYS[key][0].z;
    state.w ^= ROUND_KEYS[key][0].w;
#pragma unroll
    for (int i = 1; i < 14; i++) {
        state = {
            TBOX[0][(state.x) & 0xff] ^ TBOX[1][(state.y >> 8) & 0xff] ^ TBOX[2][(state.z >> 16) & 0xff] ^ TBOX[3][(state.w >> 24) & 0xff],
            TBOX[0][(state.y) & 0xff] ^ TBOX[1][(state.z >> 8) & 0xff] ^ TBOX[2][(state.w >> 16) & 0xff] ^ TBOX[3][(state.x >> 24) & 0xff],
            TBOX[0][(state.z) & 0xff] ^ TBOX[1][(state.w >> 8) & 0xff] ^ TBOX[2][(state.x >> 16) & 0xff] ^ TBOX[3][(state.y >> 24) & 0xff],
            TBOX[0][(state.w) & 0xff] ^ TBOX[1][(state.x >> 8) & 0xff] ^ TBOX[2][(state.y >> 16) & 0xff] ^ TBOX[3][(state.z >> 24) & 0xff],
        };
        state.x ^= ROUND_KEYS[key][i].x;
        state.y ^= ROUND_KEYS[key][i].y;
        state.z ^= ROUND_KEYS[key][i].z;
        state.w ^= ROUND_KEYS[key][i].w;
    }
    // Final round: no mixing, just SubBytes and ShiftRows
    state = {
        (TBOX[3][(state.x) & 0xff] & 0xff) ^ (TBOX[0][(state.y >> 8) & 0xff] & 0xff00) ^ (TBOX[1][(state.z >> 16) & 0xff] & 0xff0000) ^ (TBOX[2][(state.w >> 24) & 0xff] & 0xff000000),
        (TBOX[3][(state.y) & 0xff] & 0xff) ^ (TBOX[0][(state.z >> 8) & 0xff] & 0xff00) ^ (TBOX[1][(state.w >> 16) & 0xff] & 0xff0000) ^ (TBOX[2][(state.x >> 24) & 0xff] & 0xff000000),
        (TBOX[3][(state.z) & 0xff] & 0xff) ^ (TBOX[0][(state.w >> 8) & 0xff] & 0xff00) ^ (TBOX[1][(state.x >> 16) & 0xff] & 0xff0000) ^ (TBOX[2][(state.y >> 24) & 0xff] & 0xff000000),
        (TBOX[3][(state.w) & 0xff] & 0xff) ^ (TBOX[0][(state.x >> 8) & 0xff] & 0xff00) ^ (TBOX[1][(state.y >> 16) & 0xff] & 0xff0000) ^ (TBOX[2][(state.z >> 24) & 0xff] & 0xff000000),
    };
    state.x ^= ROUND_KEYS[key][14].x;
    state.y ^= ROUND_KEYS[key][14].y;
    state.z ^= ROUND_KEYS[key][14].z;
    state.w ^= ROUND_KEYS[key][14].w;
    return state;
}

// AES Keygen on CPU

bool __get_cpuid_aes() {
    unsigned int a,b,c,d;
    if (!__get_cpuid(0x1, &a, &b, &c, &d)) {
        return false;
    }
    return (bool) (c & bit_AES);
}

#define IS_ALIGNED(v, a) ((((uintptr_t) v) & ((a)-1)) == 0)

__host__ inline __m128i aes128_keyexpand(__m128i key) {
    key = _mm_xor_si128(key, _mm_slli_si128(key, 4));
    key = _mm_xor_si128(key, _mm_slli_si128(key, 4));
    return _mm_xor_si128(key, _mm_slli_si128(key, 4));
}

#define KEYEXP128_H(K1, K2, I, S) _mm_xor_si128(aes128_keyexpand(K1), \
        _mm_shuffle_epi32(_mm_aeskeygenassist_si128(K2, I), S))

#define KEYEXP256(K1, K2, I)  KEYEXP128_H(K1, K2, I, 0xff)
#define KEYEXP256_2(K1, K2) KEYEXP128_H(K1, K2, 0x00, 0xaa)

__host__ inline void aes_keygen(__m128i rk[], const void* cipherKey) {
    assert(IS_ALIGNED(cipherKey, 16));
    const void *cipherKey2 = (const char *) cipherKey + 16;

    /* 256 bit key setup */
    rk[0] = _mm_load_si128((const __m128i*) cipherKey);
    rk[1] = _mm_load_si128((const __m128i*) cipherKey2);
    rk[2] = KEYEXP256(rk[0], rk[1], 0x01);
    rk[3] = KEYEXP256_2(rk[1], rk[2]);
    rk[4] = KEYEXP256(rk[2], rk[3], 0x02);
    rk[5] = KEYEXP256_2(rk[3], rk[4]);
    rk[6] = KEYEXP256(rk[4], rk[5], 0x04);
    rk[7] = KEYEXP256_2(rk[5], rk[6]);
    rk[8] = KEYEXP256(rk[6], rk[7], 0x08);
    rk[9] = KEYEXP256_2(rk[7], rk[8]);
    rk[10] = KEYEXP256(rk[8], rk[9], 0x10);
    rk[11] = KEYEXP256_2(rk[9], rk[10]);
    rk[12] = KEYEXP256(rk[10], rk[11], 0x20);
    rk[13] = KEYEXP256_2(rk[11], rk[12]);
    rk[14] = KEYEXP256(rk[12], rk[13], 0x40);
}

#define MEM_BITS 24
#define FILTER_BITS 6
#define BUCKET_BITS 14

#define FILTER_MASK (((1 << FILTER_BITS) - 1) << BUCKET_BITS)
#define BUCKET_MASK ((1 << BUCKET_BITS) - 1)
#define NUM_BUCKETS (1 << BUCKET_BITS)
#define MEM_SIZE (1 << MEM_BITS)

struct aes_pair {
    aes_block A, B;
};

__device__ aes_pair aes[MEM_SIZE];
__device__ uint64_t nonces[MEM_SIZE];
__device__ unsigned int buckets[MEM_SIZE];

__global__ void compute_aes_kernel(uint64_t nonceStart) {
    __shared__ unsigned int next_index;
    if (threadIdx.x == 0) {
        memcpy(TBOX, CUDA_TBOX, sizeof(TBOX));
        memcpy(ROUND_KEYS, CUDA_ROUND_KEYS, sizeof(CUDA_ROUND_KEYS));
        next_index = MEM_SIZE / gridDim.x * blockIdx.x;
    }

    unsigned int last_index = MEM_SIZE / gridDim.x * (blockIdx.x + 1);

    __syncthreads();

    for (uint64_t nonce = nonceStart + blockIdx.x * blockDim.x + threadIdx.x;
            true;
            nonce += blockDim.x * gridDim.x
        ) {
        aes_block A = aes_enc(nonce, 0);
        aes_block B = aes_enc(nonce, 1);
        uint diff = A.x - B.x;
        if ((diff & FILTER_MASK) == 0) {
            unsigned int ind = atomicAdd(&next_index, 1u);
            if (ind < last_index) {
                aes[ind].A = A;
                aes[ind].B = B;
                nonces[ind] = nonce;
                buckets[ind] = __brev(diff & BUCKET_MASK) >> (32 - BUCKET_BITS);
            } else {
                return;
            }
        }
    }
}

__constant__ unsigned int difficulty;

__device__ inline aes_block add_aes_block(aes_block l, aes_block r) {
    aes_block res;
    asm ("add.cc.u32      %0, %4, %8;\n\t"
         "addc.cc.u32     %1, %5, %9;\n\t"
         "addc.cc.u32     %2, %6, %10;\n\t"
         "addc.u32        %3, %7, %11;\n\t"
         : "=r"(res.x), "=r"(res.y), "=r"(res.z), "=r"(res.w)
         : "r"(l.x), "r"(l.y), "r"(l.z), "r"(l.w),
           "r"(r.x), "r"(r.y), "r"(r.z), "r"(r.w));
    return res;
}

__device__ inline uint hamming_distance(aes_block l, aes_block r) {
    return __popc(l.x ^ r.x) + __popc(l.y ^ r.y) + __popc(l.z ^ r.z) + __popc(l.w ^ r.w);
}

__device__ unsigned int num_results = 0;
__device__ uint64_t N1, N2;

__device__ bool check_pair(const unsigned int i, const unsigned int j) {
    aes_block l = add_aes_block(aes[i].A, aes[j].B);
    aes_block r = add_aes_block(aes[i].B, aes[j].A);
    if (hamming_distance(l, r) <= 128 - difficulty) {
        // Yay we're done! Set the output
        unsigned int res_num = atomicAdd(&num_results, 1);
        if (res_num == 0) {
            N1 = nonces[i];
            N2 = nonces[j];
        }
        return true;
    }
    return false;
}

__global__ void check_pairs_kernel() {
    uint start_index = MEM_SIZE / gridDim.x * blockIdx.x;
    uint end_index = MEM_SIZE / gridDim.x * (blockIdx.x + 1);
    for (uint i = start_index + threadIdx.x; num_results == 0 && i < end_index; i += blockDim.x) {
        for (uint j = i + 1; j < end_index; j ++) {
            if (check_pair(i, j)) {
                return;
            }
        }
    }
}

template <typename T> thrust::device_ptr<T> device_ptr_symbol(const void* symbol) {
    void *tmp;
    CUDA_SAFE_CALL(hipGetSymbolAddress(&tmp, HIP_SYMBOL(symbol)));
    return thrust::device_ptr<T>((T*) tmp);
}


void go() {
    static uint64_t nonce_start = 0;
    compute_aes_kernel<<<256, 256>>>(nonce_start);
    CUDA_SAFE_CALL(hipDeviceSynchronize());
    nonce_start += (uint64_t(MEM_SIZE) << FILTER_BITS) * 4; // *4 to be conservative

    thrust::sort_by_key (
            device_ptr_symbol<unsigned int>(buckets),
            device_ptr_symbol<unsigned int>(buckets) + MEM_SIZE,
            thrust::make_zip_iterator(thrust::make_tuple(device_ptr_symbol<aes_pair>(aes), device_ptr_symbol<uint64_t>(nonces)))
    );
    CUDA_SAFE_CALL(hipDeviceSynchronize());

    check_pairs_kernel<<<256, 256>>>();
    CUDA_SAFE_CALL(hipDeviceSynchronize());
}

void parse_hex(const char s[], uint8_t v[]) {
    while (*s) {
        sscanf(s, "%2hhx", v);
        s += 2;
        v ++;
    }
}

int main(int argc, char *argv[]) {
    int major = THRUST_MAJOR_VERSION;
    int minor = THRUST_MINOR_VERSION;
    std::cout << "Thrust v" << major << "." << minor << std::endl;

    if (!__get_cpuid_aes()) {
        fprintf(stderr, "AES-NI not supported on this CPU!\n");
        return 1;
    }

    if (argc != 4) {
        printf("Usage: aesham2 SEED SEED2 DIFFICULTY\n");
        return 1;
    }

    char *seed1 = argv[1];
    char *seed2 = argv[2];

    uint8_t A[32] __attribute__((aligned(16)));
    uint8_t B[32] __attribute__((aligned(16)));
    memset(A, 0, sizeof(A));
    memset(B, 0, sizeof(B));

    parse_hex(seed1, A);
    parse_hex(seed2, B);

    __m128i __attribute__((aligned(16))) ek[2][15];
    aes_keygen(ek[0], A);
    aes_keygen(ek[1], B);
    CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(CUDA_ROUND_KEYS), ek, sizeof(ek)));

    unsigned int host_difficulty = atoi(argv[3]);
    CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(difficulty), &host_difficulty, sizeof(difficulty)));

    go();

    uint64_t nonce1, nonce2;
    CUDA_SAFE_CALL(hipMemcpyFromSymbol(&nonce1, HIP_SYMBOL(N1), sizeof(nonce1)));
    CUDA_SAFE_CALL(hipMemcpyFromSymbol(&nonce2, HIP_SYMBOL(N2), sizeof(nonce2)));
    printf("%lu %lu\n", nonce1, nonce2);
    return 0;
}

// vim: set et ts=4 sts=4 sw=4 cindent:
